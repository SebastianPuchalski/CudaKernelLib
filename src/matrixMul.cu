#include "hip/hip_runtime.h"
#include "matrixMul.cuh"

#include "hip/hip_runtime.h"
#include ""

__global__ void matrixMulKernel(float* c, float* a, float* b, int cWidth, int cHeight, int aWidth)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < cWidth && y < cHeight) {
		float sum = 0;
		for (int i = 0; i < aWidth; i++) {
			sum += a[y * aWidth + i] * b[i * cWidth + x];
		}
		c[y * cWidth + x] = sum;
	}
}

float matrixMul(float* c, float* a, float* b, int cWidth, int cHeight, int aWidth) {
	CudaBuffer<float> aBuff(aWidth * cHeight);
	CudaBuffer<float> bBuff(cWidth * aWidth);
	CudaBuffer<float> cBuff(cWidth * cHeight);

	checkCudaError(hipMemcpy(aBuff(), a, aBuff.dataSize(), hipMemcpyHostToDevice));
	checkCudaError(hipMemcpy(bBuff(), b, bBuff.dataSize(), hipMemcpyHostToDevice));

	dim3 blockSize(16, 16);
	dim3 gridSize((cWidth + blockSize.x - 1) / blockSize.x,
		          (cHeight + blockSize.y - 1) / blockSize.y);

	float elapsedTime;
	CudaEvent start, stop;
	checkCudaError(hipEventRecord(start(), 0));
	matrixMulKernel<<<gridSize, blockSize>>> (cBuff(), aBuff(), bBuff(), cWidth, cHeight, aWidth);
	checkCudaError(hipGetLastError());
	checkCudaError(hipEventRecord(stop(), 0));
	checkCudaError(hipEventSynchronize(stop()));
	checkCudaError(hipEventElapsedTime(&elapsedTime, start(), stop()));

	checkCudaError(hipDeviceSynchronize());
	checkCudaError(hipMemcpy(c, cBuff(), cBuff.dataSize(), hipMemcpyDeviceToHost));

	return elapsedTime;
}

void matrixMulRef(float* c, float* a, float* b, int cWidth, int cHeight, int aWidth) { // row-major order
	for (int y = 0; y < cHeight; y++) {
		for (int x = 0; x < cWidth; x++) {
			float sum = 0;
			for (int i = 0; i < aWidth; i++) {
				sum += a[y * aWidth + i] * b[i * cWidth + x];
			}
			c[y * cWidth + x] = sum;
		}
	}
}

void testMatrixMul(int cWidth, int cHeight, int aWidth) {
	std::vector<float> a(aWidth * cHeight);
	std::vector<float> b(cWidth * aWidth);
	fillVectorRandom(a);
	fillVectorRandom(b);

	std::vector<float> c(cWidth * cHeight);
	std::vector<float> cRef(cWidth * cHeight);
	float time = matrixMul(c.data(), a.data(), b.data(), cWidth, cHeight, aWidth);
	matrixMulRef(cRef.data(), a.data(), b.data(), cWidth, cHeight, aWidth);

	bool pass = compareVectors(c, cRef);
	std::string name = "MatrixMul(";
	name += std::to_string(cWidth) + "x";
	name += std::to_string(cHeight);
	name += ", " + std::to_string(aWidth) + ")";
	printTestItem(name, time, pass);
}

void testMatrixMul() {
	checkCudaError(hipSetDevice(0));
	testMatrixMul(512, 256, 1024);
	checkCudaError(hipDeviceReset()); // is this a right place?
}
