#include "hip/hip_runtime.h"
#include "matrixMul.cuh"

#include "hip/hip_runtime.h"
#include ""

__global__ void matrixMulKernel(float* c, float* a, float* b, int cWidth, int cHeight, int aWidth)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < cWidth && y < cHeight) {
		float sum = 0;
		for (int i = 0; i < aWidth; i++) {
			sum += a[y * aWidth + i] * b[i * cWidth + x];
		}
		c[y * cWidth + x] = sum;
	}
}

float matrixMul(CudaBuffer<float>& cHost, CudaBuffer<float>& aHost, CudaBuffer<float>& bHost,
	            int cWidth, int cHeight, int aWidth) {
	CudaBuffer<float> aDev(aWidth * cHeight, hipMemoryTypeDevice);
	CudaBuffer<float> bDev(cWidth * aWidth, hipMemoryTypeDevice);
	CudaBuffer<float> cDev(cWidth * cHeight, hipMemoryTypeDevice);
	aDev.copyFrom(aHost);
	bDev.copyFrom(bHost);

	dim3 blockSize(16, 16);
	dim3 gridSize((cWidth + blockSize.x - 1) / blockSize.x,
		          (cHeight + blockSize.y - 1) / blockSize.y);

	float elapsedTime;
	CudaEvent start, stop;
	checkCudaError(hipEventRecord(start(), 0));
	matrixMulKernel<<<gridSize, blockSize>>> (cDev(), aDev(), bDev(), cWidth, cHeight, aWidth);
	checkCudaError(hipGetLastError());
	checkCudaError(hipEventRecord(stop(), 0));
	checkCudaError(hipEventSynchronize(stop()));
	checkCudaError(hipEventElapsedTime(&elapsedTime, start(), stop()));

	checkCudaError(hipDeviceSynchronize());
	cHost.copyFrom(cDev);

	return elapsedTime;
}

void matrixMulRef(float* c, float* a, float* b, int cWidth, int cHeight, int aWidth) { // row-major order
	for (int y = 0; y < cHeight; y++) {
		for (int x = 0; x < cWidth; x++) {
			float sum = 0;
			for (int i = 0; i < aWidth; i++) {
				sum += a[y * aWidth + i] * b[i * cWidth + x];
			}
			c[y * cWidth + x] = sum;
		}
	}
}

void testMatrixMul(int cWidth, int cHeight, int aWidth) {
	CudaBuffer<float> a(aWidth * cHeight, hipMemoryTypeHost);
	CudaBuffer<float> b(cWidth * aWidth, hipMemoryTypeHost);
	a.fillWithRandom();
	b.fillWithRandom();

	CudaBuffer<float> c(cWidth * cHeight, hipMemoryTypeHost);
	CudaBuffer<float> cRef(cWidth * cHeight, hipMemoryTypeHost);
	float time = matrixMul(c, a, b, cWidth, cHeight, aWidth);
	matrixMulRef(cRef(), a(), b(), cWidth, cHeight, aWidth);

	bool pass = c.approxEqual(cRef);
	std::string name = "MatrixMul(";
	name += std::to_string(cWidth) + "x";
	name += std::to_string(cHeight);
	name += ", " + std::to_string(aWidth) + ")";
	printTestItem(name, time, pass);
}

void testMatrixMul() {
	checkCudaError(hipSetDevice(0));
	testMatrixMul(512, 256, 1024);
	checkCudaError(hipDeviceReset());
}
