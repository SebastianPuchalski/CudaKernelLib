#include "hip/hip_runtime.h"
#include "matrixMul.cuh"

#include "hip/hip_runtime.h"
#include ""

__global__ void matrixMulKernelNaive(float* c, const float* a, const float* b, int cWidth, int cHeight, int aWidth)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < cWidth && y < cHeight) {
		float sum = 0;
		for (int i = 0; i < aWidth; i++) {
			sum += a[y * aWidth + i] * b[i * cWidth + x];
		}
		c[y * cWidth + x] = sum;
	}
}

void matrixMulNaive(float* c, const float* a, const float* b, int cWidth, int cHeight, int aWidth) {
	dim3 blockSize(32, 32);
	dim3 gridSize((cWidth + blockSize.x - 1) / blockSize.x,
		(cHeight + blockSize.y - 1) / blockSize.y);
	matrixMulKernelNaive<<<gridSize, blockSize>>>(c, a, b, cWidth, cHeight, aWidth);
}

__global__ void matrixMulKernelSTiled(float* c, const float* a, const float* b, int cWidth, int aWidth)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	const int tileSize = 16;
	assert(blockDim.x == tileSize && blockDim.y == tileSize);

	__shared__ float aShared[tileSize][tileSize];
	__shared__ float bShared[tileSize][tileSize];

	float sum = 0;

	for (int i = 0; i < aWidth; i += tileSize) {
		aShared[threadIdx.y][threadIdx.x] = a[y * aWidth + (i + threadIdx.x)];
		bShared[threadIdx.y][threadIdx.x] = b[(i + threadIdx.y) * cWidth + x];
		__syncthreads();

		for (int j = 0; j < tileSize; j++) {
			sum += aShared[threadIdx.y][j] * bShared[j][threadIdx.x];
		}
		__syncthreads();
	}

	c[y * cWidth + x] = sum;
}

void matrixMulSTiled(float* c, const float* a, const float* b, int cWidth, int cHeight, int aWidth) {
	const int tileSize = 16;
	assert(cWidth % tileSize == 0);
	assert(cHeight % tileSize == 0);
	assert(aWidth % tileSize == 0);
	assert(aWidth % 128 == 0); // row alligned with 512B block
	assert(cWidth % 128 == 0); // row alligned with 512B block
	dim3 blockSize(tileSize, tileSize);
	dim3 gridSize(cWidth / tileSize, cHeight / tileSize);
	matrixMulKernelSTiled<<<gridSize, blockSize>>>(c, a, b, cWidth, aWidth);
}

__global__ void matrixMulKernelSTTiled(float* c, const float* a, const float* b, int cWidth, int aWidth)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	const int tileSize = 4;
	const int blockSize = 16;
	assert(blockDim.x == blockSize && blockDim.y == blockSize);

	__shared__ float4 aShared[blockSize * tileSize * blockSize];
	__shared__ float4 bShared[blockSize * tileSize * blockSize];

	float4 row0 = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	float4 row1 = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	float4 row2 = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	float4 row3 = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

	for (int i = 0; i < aWidth; i += blockSize * tileSize) {
		#pragma unroll
		for (int j = 0; j < tileSize; j++) {
			aShared[(threadIdx.y * tileSize + j) * blockSize + threadIdx.x] =
				*((const float4*)(a + ((y * tileSize + j) * aWidth + (i + threadIdx.x * tileSize))));
			bShared[(threadIdx.y * tileSize + j) * blockSize + threadIdx.x] =
				*((const float4*)(b + ((i + threadIdx.y * tileSize + j) * cWidth + (x * tileSize))));
		}
		__syncthreads();

		float* aS = (float*)aShared + threadIdx.y * tileSize * blockSize * tileSize;
		float* bS = (float*)bShared + (threadIdx.x * tileSize);

        #pragma unroll
		for (int j = 0; j < blockSize * tileSize; j++) {
			float a0 = aS[0 * blockSize * tileSize + j];
			float a1 = aS[1 * blockSize * tileSize + j];
			float a2 = aS[2 * blockSize * tileSize + j];
			float a3 = aS[3 * blockSize * tileSize + j];
			float4 b = *((float4*)(bS + j * blockSize * tileSize));

			row0.x += a0 * b.x;
			row0.y += a0 * b.y;
			row0.z += a0 * b.z;
			row0.w += a0 * b.w;

			row1.x += a1 * b.x;
			row1.y += a1 * b.y;
			row1.z += a1 * b.z;
			row1.w += a1 * b.w;

			row2.x += a2 * b.x;
			row2.y += a2 * b.y;
			row2.z += a2 * b.z;
			row2.w += a2 * b.w;

			row3.x += a3 * b.x;
			row3.y += a3 * b.y;
			row3.z += a3 * b.z;
			row3.w += a3 * b.w;
		}
		__syncthreads();
	}

	c = (c + y * tileSize * cWidth) + x * tileSize;
	*((float4*)c) = row0;
	c += cWidth;
	*((float4*)c) = row1;
	c += cWidth;
	*((float4*)c) = row2;
	c += cWidth;
	*((float4*)c) = row3;
}

void matrixMulSTTiled(float* c, const float* a, const float* b, int cWidth, int cHeight, int aWidth) {
	const int tileSize = 4;
	assert(cWidth % tileSize == 0);
	assert(cHeight % tileSize == 0);
	assert(aWidth % tileSize == 0);
	assert(aWidth % 128 == 0); // row alligned with 512B block
	assert(cWidth % 128 == 0); // row alligned with 512B block
	dim3 blockSize(16, 16);
	dim3 matPartSize(blockSize.x * tileSize, blockSize.y * tileSize);
	assert(cWidth % matPartSize.x == 0);
	assert(cHeight % matPartSize.y == 0);
	dim3 gridSize(cWidth / matPartSize.x, cHeight / matPartSize.y);
	matrixMulKernelSTTiled<<<gridSize, blockSize>>>(c, a, b, cWidth, aWidth);
}

template <int width, int height, int threadCount>
__device__ void loadRect(float* dst, const float* src, int srcStride, int thread) {
	const int vecSize = sizeof(float4) / sizeof(float);
	assert(width % vecSize == 0);
	const int w = width / vecSize;
	assert(width / vecSize <= threadCount);
	const int h = threadCount / w;
	assert(threadCount % w == 0);

	int ty = thread / w;
	int tx = thread - (ty * w);
	tx *= vecSize;
	src += tx;
	dst += tx;

    #pragma unroll
	for (int i = ty; i < height; i += h) {
		float4 vec = *reinterpret_cast<const float4*>(src + i * srcStride);
		reinterpret_cast<float4*>(dst)[i * w] = vec;
	}
}

__global__ void matrixMulKernelSWTiled(float* c, const float* a, const float* b, int cWidth, int aWidth)
{
	const int sharedSize = 32;
	const int tileWidth = 32;
	const int tileHeight = 64;
	const int gridWidth = 4;
	const int gridHeight = 2;
	assert(blockDim.x == tileWidth);
	assert(blockDim.y == gridWidth && blockDim.z == gridHeight);
	const int threadCount = tileWidth * gridWidth * gridHeight;

	int warpThread = threadIdx.x;
	int tileX = threadIdx.y;
	int tileY = threadIdx.z;
	int thread = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
	int x = blockIdx.x * blockDim.y * tileWidth;
	int y = blockIdx.y * blockDim.z * tileHeight;

	__shared__ float aS[gridHeight * tileHeight * sharedSize];
	__shared__ float bS[sharedSize * gridWidth * tileWidth];

	float accum[tileHeight];
	for (int i = 0; i < tileHeight; i++)
		accum[i] = 0;

	a += y * aWidth;
	b += x;

	for (int i = 0; i < aWidth; i += sharedSize) {
		loadRect<sharedSize, gridHeight * tileHeight, threadCount>((float*)aS, a + i, aWidth, thread);
		loadRect<gridWidth * tileWidth, sharedSize, threadCount>((float*)bS, b + i * cWidth, cWidth, thread);
		__syncthreads();

		float* a = aS + tileY * tileHeight * sharedSize;
		float* b = bS + tileX * tileWidth + warpThread;
        #pragma unroll
		for (int k = 0; k < sharedSize; k++) {
			float bValue = b[k * gridWidth * tileWidth];
			#pragma unroll
			for (int j = 0; j < tileHeight; j++) {
				accum[j] += a[j * sharedSize + k] * bValue;
			}
		}
		__syncthreads();
	}

	x += tileX * tileWidth;
	y += tileY * tileHeight;
	c += x + warpThread;
	c += y * cWidth;
	#pragma unroll
	for (int i = 0; i < tileHeight; i++)
		c[i * cWidth] = accum[i];
}

void matrixMulSWTiled(float* c, const float* a, const float* b, int cWidth, int cHeight, int aWidth) {
	const int sharedSize = 32;
	const int tileWidth = 32;
	const int tileHeight = 64;
	assert(tileWidth == 32); // tileWidth has to have warp size
	assert(aWidth % 128 == 0); // row alligned with 512B block
	assert(cWidth % 128 == 0); // row alligned with 512B block
	dim3 blockSize(32, 4, 2);
	dim3 accumSize(blockSize.y * tileWidth, blockSize.z * tileHeight);
	assert(aWidth % sharedSize == 0);
	assert(cWidth % accumSize.x == 0);
	assert(cHeight % accumSize.y == 0);
	dim3 gridSize(cWidth / accumSize.x, cHeight / accumSize.y);
	matrixMulKernelSWTiled<<<gridSize, blockSize>>>(c, a, b, cWidth, aWidth);
}

template <int width, int height, int threadCount>
__device__ void loadRectT(float* dst, const float* src, int srcStride, int thread) {
	const int vecSize = sizeof(float4) / sizeof(float);
	assert(width % vecSize == 0);
	if (threadCount == height * 2) {
		int ty = thread >> 1;
		int tx = thread - ty * 2;
		const float4* srcRow = reinterpret_cast<const float4*>(src + ty * srcStride);
		dst += ty;

		#pragma unroll
		for (int x = tx; x < width / vecSize; x += 2) {
			float4 vec = srcRow[x];
			dst[(x * vecSize + 0) * height] = vec.x;
			dst[(x * vecSize + 1) * height] = vec.y;
			dst[(x * vecSize + 2) * height] = vec.z;
			dst[(x * vecSize + 3) * height] = vec.w;
		}
	}
	else {
		assert(threadCount % 2 == 0);
		const int threadBlockH = min(threadCount / 2, height);
		assert(threadCount % threadBlockH == 0);
		const int threadBlockW = threadCount / threadBlockH;
		assert((width / vecSize) % threadBlockW == 0);
		assert(height % threadBlockH == 0);

		int ty = thread / threadBlockW;
		int tx = thread - (ty * threadBlockW);

		#pragma unroll
		for (int y = ty; y < height; y += threadBlockH) {
			const float4* srcRow = reinterpret_cast<const float4*>(src + y * srcStride);
			float* dstCol = dst + y;
			#pragma unroll
			for (int x = tx; x < width / vecSize; x += threadBlockW) {
				float4 vec = srcRow[x];
				dstCol[(x * vecSize + 0) * height] = vec.x;
				dstCol[(x * vecSize + 1) * height] = vec.y;
				dstCol[(x * vecSize + 2) * height] = vec.z;
				dstCol[(x * vecSize + 3) * height] = vec.w;
			}
		}
	}
}

template <int sharedSize, int tileWidth, int tileHeight, int gridWidth, int gridHeight>
__global__ void matrixMulKernelFast(float* c, const float* a, const float* b, int cWidth, int aWidth)
{
	const int warpSize = 32;
	const int vecSize = sizeof(float4) / sizeof(float);
	const int accumWidth = gridWidth * tileWidth;
	const int accumHeight = gridHeight * tileHeight;
	const int threadCount = warpSize * gridWidth * gridHeight;
	const int warpWidth = tileWidth / vecSize / 2;
	const int warpHeight = tileHeight / vecSize / 2;

	assert(blockDim.x == warpSize);
	assert(blockDim.y == gridWidth && blockDim.z == gridHeight);
	assert(warpWidth * warpHeight == warpSize);

	int warpThread = threadIdx.x;
	int tileX = threadIdx.y;
	int tileY = threadIdx.z;
	int threadX = warpThread % warpWidth;
	int threadY = warpThread / warpWidth;
	int thread = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
	int x = blockIdx.x * blockDim.y * tileWidth;
	int y = blockIdx.y * blockDim.z * tileHeight;

	__shared__ float aS[sharedSize * accumHeight];
	__shared__ float bS[sharedSize * accumWidth];

	float4 accum00[vecSize];
	float4 accum01[vecSize];
	float4 accum10[vecSize];
	float4 accum11[vecSize];
    #pragma unroll
	for (int i = 0; i < vecSize; i++) {
		accum00[i] = accum01[i] = accum10[i] = accum11[i] = make_float4(0.f, 0.f, 0.f, 0.f);
	}

	a += y * aWidth;
	b += x;

	for (int i = 0; i < aWidth; i += sharedSize) {
		loadRectT<sharedSize, gridHeight * tileHeight, threadCount>((float*)aS, a + i, aWidth, thread);
		loadRect<gridWidth * tileWidth, sharedSize, threadCount>((float*)bS, b + i * cWidth, cWidth, thread);
		__syncthreads();

		float4* b = reinterpret_cast<float4*>(bS + tileX * tileWidth) + threadX;
		float4* a = reinterpret_cast<float4*>(aS + tileY * tileHeight) + threadY;
		const int bStride = accumWidth / vecSize;
		const int aStride = accumHeight / vecSize;

        #pragma unroll
		for (int j = 0; j < sharedSize; j++) {
			float4 b0, b1;
			b0 = b[j * bStride];
			b1 = b[j * bStride + warpWidth];
			float4 a0, a1;
			a0 = a[j * aStride];
			a1 = a[j * aStride + warpHeight];

			accum00[0].x += a0.x * b0.x; accum00[0].y += a0.x * b0.y; accum00[0].z += a0.x * b0.z; accum00[0].w += a0.x * b0.w;
			accum00[1].x += a0.y * b0.x; accum00[1].y += a0.y * b0.y; accum00[1].z += a0.y * b0.z; accum00[1].w += a0.y * b0.w;
			accum00[2].x += a0.z * b0.x; accum00[2].y += a0.z * b0.y; accum00[2].z += a0.z * b0.z; accum00[2].w += a0.z * b0.w;
			accum00[3].x += a0.w * b0.x; accum00[3].y += a0.w * b0.y; accum00[3].z += a0.w * b0.z; accum00[3].w += a0.w * b0.w;

			accum01[0].x += a0.x * b1.x; accum01[0].y += a0.x * b1.y; accum01[0].z += a0.x * b1.z; accum01[0].w += a0.x * b1.w;
			accum01[1].x += a0.y * b1.x; accum01[1].y += a0.y * b1.y; accum01[1].z += a0.y * b1.z; accum01[1].w += a0.y * b1.w;
			accum01[2].x += a0.z * b1.x; accum01[2].y += a0.z * b1.y; accum01[2].z += a0.z * b1.z; accum01[2].w += a0.z * b1.w;
			accum01[3].x += a0.w * b1.x; accum01[3].y += a0.w * b1.y; accum01[3].z += a0.w * b1.z; accum01[3].w += a0.w * b1.w;

			accum10[0].x += a1.x * b0.x; accum10[0].y += a1.x * b0.y; accum10[0].z += a1.x * b0.z; accum10[0].w += a1.x * b0.w;
			accum10[1].x += a1.y * b0.x; accum10[1].y += a1.y * b0.y; accum10[1].z += a1.y * b0.z; accum10[1].w += a1.y * b0.w;
			accum10[2].x += a1.z * b0.x; accum10[2].y += a1.z * b0.y; accum10[2].z += a1.z * b0.z; accum10[2].w += a1.z * b0.w;
			accum10[3].x += a1.w * b0.x; accum10[3].y += a1.w * b0.y; accum10[3].z += a1.w * b0.z; accum10[3].w += a1.w * b0.w;

			accum11[0].x += a1.x * b1.x; accum11[0].y += a1.x * b1.y; accum11[0].z += a1.x * b1.z; accum11[0].w += a1.x * b1.w;
			accum11[1].x += a1.y * b1.x; accum11[1].y += a1.y * b1.y; accum11[1].z += a1.y * b1.z; accum11[1].w += a1.y * b1.w;
			accum11[2].x += a1.z * b1.x; accum11[2].y += a1.z * b1.y; accum11[2].z += a1.z * b1.z; accum11[2].w += a1.z * b1.w;
			accum11[3].x += a1.w * b1.x; accum11[3].y += a1.w * b1.y; accum11[3].z += a1.w * b1.z; accum11[3].w += a1.w * b1.w;
		}
		__syncthreads();
	}

	x += tileX * tileWidth + threadX * vecSize;
	y += tileY * tileHeight + threadY * vecSize;
#define STORE_OPT
#ifdef STORE_OPT
	c += y * cWidth + x;
    #pragma unroll
	for (int i = 0; i < vecSize; i++) {
		c[i * cWidth + 0] = accum00[i].x;
		c[i * cWidth + 1] = accum00[i].y;
		c[i * cWidth + 2] = accum00[i].z;
		c[i * cWidth + 3] = accum00[i].w;
	}
	for (int i = 0; i < vecSize; i++) {
		c[i * cWidth + vecSize * warpWidth + 0] = accum01[i].x;
		c[i * cWidth + vecSize * warpWidth + 1] = accum01[i].y;
		c[i * cWidth + vecSize * warpWidth + 2] = accum01[i].z;
		c[i * cWidth + vecSize * warpWidth + 3] = accum01[i].w;
	}
	for (int i = 0; i < vecSize; i++) {
		c[i * cWidth + vecSize * warpHeight * cWidth + 0] = accum10[i].x;
		c[i * cWidth + vecSize * warpHeight * cWidth + 1] = accum10[i].y;
		c[i * cWidth + vecSize * warpHeight * cWidth + 2] = accum10[i].z;
		c[i * cWidth + vecSize * warpHeight * cWidth + 3] = accum10[i].w;
	}
	for (int i = 0; i < vecSize; i++) {
		c[i * cWidth + vecSize * (warpHeight * cWidth + warpWidth) + 0] = accum11[i].x;
		c[i * cWidth + vecSize * (warpHeight * cWidth + warpWidth) + 1] = accum11[i].y;
		c[i * cWidth + vecSize * (warpHeight * cWidth + warpWidth) + 2] = accum11[i].z;
		c[i * cWidth + vecSize * (warpHeight * cWidth + warpWidth) + 3] = accum11[i].w;
	}
#else
	float4* out = reinterpret_cast<float4*>(c + y * cWidth + x);
	int outStride = cWidth >> 2;
	#pragma unroll
	for (int i = 0; i < vecSize; i++) {
		out[i * outStride] = accum00[i];
		out[i * outStride + warpWidth] = accum01[i];
		out[i * outStride + warpHeight * cWidth] = accum10[i];
		out[i * outStride + warpHeight * cWidth + warpWidth] = accum11[i];
	}
#endif
}

void matrixMulFast(float* c, const float* a, const float* b, int cWidth, int cHeight, int aWidth) {
	assert(aWidth % 128 == 0); // row alligned with 512B block
	assert(cWidth % 128 == 0); // row alligned with 512B block
	const int warpSize = 32;
	const int sharedSize = 32;
	const int tileWidth = 64;
	const int tileHeight = 32;
	const int gridWidth = 2;
	const int gridHeight = 2;
	dim3 blockSize(warpSize, gridWidth, gridHeight);
	dim3 accumSize(gridWidth * tileWidth, gridHeight * tileHeight);
	assert(aWidth % sharedSize == 0);
	assert(cWidth % accumSize.x == 0);
	assert(cHeight % accumSize.y == 0);
	dim3 gridSize(cWidth / accumSize.x, cHeight / accumSize.y);
	matrixMulKernelFast<sharedSize, tileWidth, tileHeight, gridWidth, gridHeight>
		<<<gridSize, blockSize>>>(c, a, b, cWidth, aWidth);
}

using KernelFunction = void(*)(float*, const float*, const float*, int, int, int);

float matrixMul(CudaBuffer<float>& cHost, const CudaBuffer<float>& aHost, const CudaBuffer<float>& bHost,
	            int cWidth, int cHeight, int aWidth, KernelFunction kernelFunc) {
	CudaBuffer<float> cDev(cHost.size(), hipMemoryTypeDevice);
	CudaBuffer<float> aDev(aHost.size(), hipMemoryTypeDevice);
	CudaBuffer<float> bDev(bHost.size(), hipMemoryTypeDevice);
	aDev.copyFrom(aHost);
	bDev.copyFrom(bHost);

	CudaEvent start, stop;
	start.record();
	kernelFunc(cDev(), aDev(), bDev(), cWidth, cHeight, aWidth);
	checkCudaError(hipGetLastError());
	stop.record();
	stop.synchronize();
	float elapsedTime = start.elapsedTime(stop);

	checkCudaError(hipDeviceSynchronize());
	cHost.copyFrom(cDev);

	return elapsedTime;
}

void matrixMulRef(float* c, const float* a, const float* b,
	int cWidth, int cHeight, int aWidth, bool tiled = true) { // row-major order
	if (!tiled) {
		for (int y = 0; y < cHeight; y++) {
			for (int x = 0; x < cWidth; x++) {
				float sum = 0;
				for (int i = 0; i < aWidth; i++) {
					sum += a[y * aWidth + i] * b[i * cWidth + x];
				}
				c[y * cWidth + x] = sum;
			}
		}
	}
	else {
		const int tileSize = 128;
		float accum[tileSize][tileSize];
		for (int ty = 0; ty < cHeight; ty += tileSize) {
			for (int tx = 0; tx < cWidth; tx += tileSize) {
				int tileWidth = std::min(cWidth - tx, tileSize);
				int tileHeight = std::min(cHeight - ty, tileSize);
				for (int y = 0; y < tileHeight; y++) {
					for (int x = 0; x < tileWidth; x++) {
						accum[y][x] = 0;
					}
				}
				for (int i = 0; i < aWidth; i++) {
					const float* aPtr = a + ty * aWidth + i;
					const float* bPtr = b + i * cWidth + tx;
					for (int y = 0; y < tileHeight; y++) {
						float aValue = aPtr[y * aWidth];
						for (int x = 0; x < tileWidth; x++) {
							accum[y][x] += aValue * bPtr[x];
						}
					}
				}
				for (int y = 0; y < tileHeight; y++) {
					for (int x = 0; x < tileWidth; x++) {
						c[(ty + y) * cWidth + (tx + x)] = accum[y][x];
					}
				}
			}
		}
	}
}

void testMatrixMul(int cWidth, int cHeight, int aWidth,
	               KernelFunction kernelFunc, const std::string& kernelName = "") {
	assert(cWidth > 0 && cHeight > 0 && aWidth > 0);
	CudaBuffer<float> a(aWidth * cHeight, hipMemoryTypeHost);
	CudaBuffer<float> b(cWidth * aWidth, hipMemoryTypeHost);
	a.fillWithRandom();
	b.fillWithRandom();

	CudaBuffer<float> c(cWidth * cHeight, hipMemoryTypeHost);
	float time = matrixMul(c, a, b, cWidth, cHeight, aWidth, kernelFunc);
	CudaBuffer<float> cRef(cWidth * cHeight, hipMemoryTypeHost);
	matrixMulRef(cRef(), a(), b(), cWidth, cHeight, aWidth);

	bool pass = c.approxEqual(cRef);
	std::string name = "MatrixMul";
	name += kernelName + "(";
	name += std::to_string(cWidth) + "x";
	name += std::to_string(cHeight);
	name += ", " + std::to_string(aWidth) + ")";
	printTestItem(name, time, pass);
}

void testMatrixMul() {
	checkCudaError(hipSetDevice(0));

	testMatrixMul(1024, 1024, 1024, matrixMulNaive, "Naive");
	testMatrixMul(1024, 1024, 1024, matrixMulSTiled, "STiled");
	testMatrixMul(1024, 1024, 1024, matrixMulSTTiled, "STTiled");
	testMatrixMul(1024, 1024, 1024, matrixMulSWTiled, "SWTiled");
	testMatrixMul(1024, 1024, 1024, matrixMulFast, "Fast");

	checkCudaError(hipGetLastError());
	checkCudaError(hipDeviceReset());
}
